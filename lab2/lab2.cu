#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include "hip/hip_runtime.h"


#define CSC(call)                                                   \
do {                                                                \
    hipError_t res = call;                                         \
    if (res != hipSuccess) {                                       \
        fprintf(stderr, "ERROR in %s:%d. Message: %s\n",            \
                __FILE__, __LINE__, hipGetErrorString(res));       \
        exit(0);                                                    \
    }                                                               \
} while(0)


__global__ void kernel(hipTextureObject_t texObj, uchar4 *out, int w, int h) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int idy = blockDim.y * blockIdx.y + threadIdx.y;
   	int offsetx = blockDim.x * gridDim.x;
	int offsety = blockDim.y * gridDim.y;
    int x, y;
    uchar4 p;
    int kernelX[3][3] = {
        {-1, 0, 1},
        {-2, 0, 2},
        {-1, 0, 1}
    };
    int kernelY[3][3] = {
        {-1, -2, -1},
        { 0,  0,  0},
        { 1,  2,  1}
    };

    for (y = idy; y < h; y += offsety) {
        for (x = idx; x < w; x += offsetx) {
            float Gx = 0.0f, Gy = 0.0f;

            for (int i = -1; i <= 1; i++) {
                for (int j = -1; j <= 1; j++) {
                    p = tex2D<uchar4>(texObj, x + j, y + i);

                    float YUV = 0.2989f * p.x + 0.587f * p.y + 0.114f * p.z;
                    Gx += YUV * kernelX[i + 1][j + 1];
                    Gy += YUV * kernelY[i + 1][j + 1];
                }
            }

            float f = sqrtf(Gx * Gx + Gy * Gy);
            if (f > 255.0f) {
                f = 255.0f;
            }
            out[y * w + x] = make_uchar4(
                static_cast<u_char>(f), static_cast<u_char>(f), static_cast<u_char>(f), 255
            );
        }
    }
}

int main() {
    char inputFileName[PATH_MAX];
    scanf("%s", inputFileName);
    char outputFileName[PATH_MAX];
    scanf("%s", outputFileName);

    int w, h;
    FILE *fp = fopen(inputFileName, "rb");
    fread(&w, sizeof(int), 1, fp);
    fread(&h, sizeof(int), 1, fp);
    uchar4 *data = (uchar4 *)malloc(sizeof(uchar4) * w * h);
    fread(data, sizeof(uchar4), w * h, fp);
    fclose(fp);

    hipArray *arr;
    hipChannelFormatDesc ch = hipCreateChannelDesc<uchar4>();
    CSC(hipMallocArray(&arr, &ch, w, h));
    CSC(hipMemcpy2DToArray(arr, 0, 0, data, w * sizeof(uchar4), w * sizeof(uchar4), h, hipMemcpyHostToDevice));

    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = arr;

    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModePoint;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = false;

    hipTextureObject_t tex = 0;
    CSC(hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL));

    uchar4 *dev_out;
    CSC(hipMalloc(&dev_out, sizeof(uchar4) * w * h));

    kernel<<<dim3(16, 16), dim3(32, 32)>>>(tex, dev_out, w, h);
    CSC(hipGetLastError());

    CSC(hipMemcpy(data, dev_out, sizeof(uchar4) * w * h, hipMemcpyDeviceToHost));

    CSC(hipDestroyTextureObject(tex));
    CSC(hipFreeArray(arr));
    CSC(hipFree(dev_out));

    fp = fopen(outputFileName, "wb");
    fwrite(&w, sizeof(int), 1, fp);
    fwrite(&h, sizeof(int), 1, fp);
    fwrite(data, sizeof(uchar4), w * h, fp);
    fclose(fp);

    free(data);
    return 0;
}
